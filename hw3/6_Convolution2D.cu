#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>

// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

#include <time.h>
#include "gputimer.h"

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005



////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter,
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// GPU: Row convolution Kernel
////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_rows(const double *filter, const double *input, double *output,
                       int imageW, int imageH, int filterR){
  int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
  int idx_y = threadIdx.y + blockDim.y * blockIdx.y;

  int grid_width = gridDim.x * blockDim.x;
  int idx = grid_width * idx_y + idx_x;

  double sum = 0;
  int k;

  // Rows
  for(k = -filterR; k <= filterR; k++){
    int d = idx_x + k;

    if(d >= 0 && d < imageW){
      sum += input[idx_y * imageW + d] * filter[filterR - k];
    }
  }

  output[idx] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// GPU: Column convolution Kernel
////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_columns(const double *filter, const double *buffer, double *output,
                       int imageW, int imageH, int filterR){
  int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
  int idx_y = threadIdx.y + blockDim.y * blockIdx.y;

  int grid_width = gridDim.x * blockDim.x;
  int idx = grid_width * idx_y + idx_x;

  double sum = 0;
  int k;

  // Columns
  for(k = -filterR; k <= filterR; k++){
    int d = idx_y + k;

    if(d >= 0 && d < imageH){
      sum += buffer[d * imageW + idx_x] * filter[filterR - k];
    }
  }

  output[idx] = sum;
}

// Auxiliary function for CUDA error checking
void cudaCheckForErrors(){
  hipError_t error = hipGetLastError();
  if(error != hipSuccess){
    // something's gone wrong
    // print out the CUDA error as a string
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    exit(1);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(void) {

    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU;

    // GPU
    double *d_Filter, *d_Input, *d_Buffer, *d_OutputGPU, *h_OutputGPU;

    unsigned int imageW;
    unsigned int imageH;
    unsigned int i;

    GpuTimer timer;
    clock_t start_CPU, end_CPU;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));


    hipMalloc( (void **) &d_Filter,      FILTER_LENGTH * sizeof(double));
    hipMalloc( (void **) &d_Input,     imageW * imageH * sizeof(double));
    hipMalloc( (void **) &d_Buffer,     imageW * imageH * sizeof(double));
    hipMalloc( (void **) &d_OutputGPU, imageW * imageH * sizeof(double));

    if(!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU || !h_OutputGPU){
        printf("error allocating memory for the host\n");
        exit(1);
   }

    if(!d_Filter || !d_Input || !d_Buffer || !d_OutputGPU){
      printf("Error allocating memory for the device\n");
      exit(1);
    }

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }

    //////////////////////////////// CPU ///////////////////////////////////////
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    start_CPU = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    end_CPU = clock();
    printf("CPU Time: %lf ms\n", ((double) ((end_CPU - start_CPU) * 1000)) / CLOCKS_PER_SEC);

    //////////////////////////////// GPU ///////////////////////////////////////
    dim3 block_dim;
    dim3 grid_dim;

    if(imageW < 32){
      block_dim.x = imageW;
      block_dim.y = imageH;

      grid_dim.x = 1;
      grid_dim.y = 1;

    } else{
      block_dim.x = 32;
      block_dim.y = 32;

      grid_dim.x = imageW / block_dim.x;
      grid_dim.y = imageH / block_dim.y;
}

    printf("GPU computation...\n");

    timer.Start();

    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(double), hipMemcpyHostToDevice);

    kernel_rows<<<grid_dim, block_dim>>>(d_Filter, d_Input, d_Buffer, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    cudaCheckForErrors();

    kernel_columns<<<grid_dim, block_dim>>>(d_Filter, d_Buffer, d_OutputGPU, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    cudaCheckForErrors();

    hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(double), hipMemcpyDeviceToHost);

    timer.Stop();
    printf("GPU Time elapsed = %lf ms\n", timer.Elapsed());

    //////////////////////// RESULT COMPARISON /////////////////////////////////

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

    for(i = 0; i < imageH * imageW; i++){
      if(ABS(h_OutputGPU[i] - h_OutputCPU[i]) >= accuracy){
        printf("GPU computations are not as accurate as we want.\n");
        break;
      }
    }

    ////////////////// CPU: free all the allocated memory //////////////////////
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    ////////////////// GPU: free all the allocated memory //////////////////////
    free(h_OutputGPU);
    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
