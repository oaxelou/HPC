/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// #include <cuda.h>
// #include <cuda_runtime_api.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005



////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter,
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// GPU: Row convolution Kernel
////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_rows(const float *filter, const float *input, float *output,
                       int imageW, int imageH, int filterR){
  int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
  int idx_y = threadIdx.y + blockDim.y * blockIdx.y;

  int grid_width = gridDim.x * blockDim.x;
  int idx = grid_width * idx_y + idx_x;

  float sum = 0;
  int k;

  // Rows
  for(k = -filterR; k <= filterR; k++){
    int d = idx_x + k;

    if(d >= 0 && d < imageW){
      sum += input[idx_y * imageW + d] * filter[filterR - k];
    }
  }

  output[idx] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// GPU: Column convolution Kernel
////////////////////////////////////////////////////////////////////////////////
__global__ void kernel_columns(const float *filter, const float *buffer, float *output,
                       int imageW, int imageH, int filterR){
  int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
  int idx_y = threadIdx.y + blockDim.y * blockIdx.y;

  int grid_width = gridDim.x * blockDim.x;
  int idx = grid_width * idx_y + idx_x;

  float sum = 0;
  int k;

  // Columns
  for(k = -filterR; k <= filterR; k++){
    int d = idx_y + k;

    if(d >= 0 && d < imageH){
      sum += buffer[d * imageW + idx_x] * filter[filterR - k];
    }
  }

  output[idx] = sum;
}


// Auxiliary function for CUDA error checking
void cudaCheckForErrors(){
  hipError_t error = hipGetLastError();
  if(error != hipSuccess){
    // something's gone wrong
    // print out the CUDA error as a string
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    exit(1);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(void) {

    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU;

    // GPU
    float *d_Filter, *d_Input, *d_Buffer, *d_OutputGPU, *h_OutputGPU;

    unsigned int imageW;
    unsigned int imageH;
    unsigned int i;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));


    hipMalloc( (void **) &d_Filter,      FILTER_LENGTH * sizeof(float));
    hipMalloc( (void **) &d_Input,     imageW * imageH * sizeof(float));
    hipMalloc( (void **) &d_Buffer,     imageW * imageH * sizeof(float));
    hipMalloc( (void **) &d_OutputGPU, imageW * imageH * sizeof(float));

    if(!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU || !h_OutputGPU){
        printf("error allocating memory for the host\n");
        exit(1);
   }

    if(!d_Filter || !d_Input || !d_Buffer || !d_OutputGPU){
      printf("Error allocating memory for the device\n");
      exit(1);
    }

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }

    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);

    //////////////////////////////// CPU ///////////////////////////////////////
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles


    //////////////////////////////// GPU ///////////////////////////////////////
    dim3 block_dim;
    block_dim.x = imageW;
    block_dim.y = imageH;

    dim3 grid_dim;
    grid_dim.x = 1;
    grid_dim.y = 1;

    printf("GPU computation...\n");

    kernel_rows<<<grid_dim, block_dim>>>(d_Filter, d_Input, d_Buffer, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    cudaCheckForErrors();

    kernel_columns<<<grid_dim, block_dim>>>(d_Filter, d_Buffer, d_OutputGPU, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    cudaCheckForErrors();

    hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);

    //////////////////////// RESULT COMPARISON /////////////////////////////////

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

    for(i = 0; i < imageH * imageW; i++){
      if(ABS(h_OutputGPU[i] - h_OutputCPU[i]) >= accuracy){
        printf("GPU computations are not as accurate as we want.\n");
        break;
      }
    }

    ////////////////// CPU: free all the allocated memory //////////////////////
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    ////////////////// GPU: free all the allocated memory //////////////////////
    free(h_OutputGPU);
    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
