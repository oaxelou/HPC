/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005



////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter,
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}

__global__ void kernel(const float *filter, const float *input, float *output){
  int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
  int idx_y = threadIdx.y + blockDim.y * blockIdx.y;

  int grid_width = gridDim.x * blockDim.x;
  int idx = grid_width * idx_y + idx_x;

  // printf("I am: idx_y %d + %d * %d = %d / idx_x %d + %d * %d = %d. Generally: %d\n",
  // threadIdx.y, blockDim.y, blockIdx.y, idx_y,
  // threadIdx.x, blockDim.x, blockIdx.x, idx_x, idx);
  output[idx] = input[idx] + 1;
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(void) {

    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU;

    // GPU
    float *d_Filter, *d_Input, *d_OutputGPU, *h_OutputGPU;

    int imageW;
    int imageH;
    unsigned int i;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));


    hipMalloc( (void **) &d_Filter,      FILTER_LENGTH * sizeof(float));
    hipMalloc( (void **) &d_Input,     imageW * imageH * sizeof(float));
    hipMalloc( (void **) &d_OutputGPU, imageW * imageH * sizeof(float));

    if(!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU ||
       !d_Filter || !d_Input || !d_OutputGPU || !h_OutputGPU){
      printf("Error allocating memory.\n");
      exit(1);
    }

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }

    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);

    printf("Input:\n");
    for(int i = 0; i < imageH; i++){
      for(int j = 0; j < imageW; j++)
        printf("%g ", h_Input[i * imageW + j]);
      printf("\n");
    }

    // ********************* CPU *********************
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    // printf("Results CPU:\n");
    // for(int i = 0; i < imageH; i++){
    //   for(int j = 0; j < imageW; j++)
    //     printf("%g ", h_OutputCPU[i * imageW + j]);
    //   printf("\n");
    // }

    //******************* GPU ************************
    // grid_dim, block_dim

    dim3 block_dim;
    block_dim.x = 4;
    block_dim.y = 4;

    dim3 grid_dim;
    grid_dim.x = imageW / block_dim.x;
    grid_dim.y = imageH / block_dim.x;

    // if((imageW * imageH) % block_size) ++grid_size;  // xreiazetai ?!

    kernel<<<grid_dim, block_dim>>>(d_Filter, d_Input, d_OutputGPU);
    hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);

    printf("Results GPU:\n");
    for(int i = 0; i < imageH; i++){
      for(int j = 0; j < imageW; j++)
        printf("%g ", h_OutputGPU[i * imageW + j]);
      printf("\n");
    }

    //************ RESULT COMPARISON *****************

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

    // for(int i = 0; i < imageH * imageW; i++){
    //   if(ABS(h_OutputGPU[i] - h_OutputCPU[i]) >= accuracy){
    //     printf("ERROR with CUDA. Something went wrong.\n");
    //     break;
    //   }
    // }

    //********** CPU *************
    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    //********** GPU *************
    free(h_OutputGPU);
    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
